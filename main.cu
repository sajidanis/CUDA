#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "marketReader.cuh"
#include "mis.cuh"
#include "scc.cuh"

using namespace std;

int main(int argc, char **argv){
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <market_file>" << " <GPU_ID>" << std::endl;
        return EXIT_FAILURE;
    }

    std::string filename = argv[1];
    int gpuId = atoi(argv[2]);

    COO_t coo;     
    CSR_t csr;
    ReadMarketStream(filename, coo);
    COO_to_CSR(coo, csr);

    size_t numNodes = csr.nodes;
    // size_t numEdges = csr.edges;

    thrust::device_vector<int> d_offsets(csr.offsets);
    thrust::device_vector<int> d_colIndices(csr.columnIndices);

    for(auto &el : csr.offsets){
        cout << el << ", ";
    }
    cout << "\n";

    for(auto &el : csr.columnIndices){
        cout << el << ",";
    }
    cout << "\n";

    // thrust::device_vector<int> d_MIS(numNodes, -1);
    // Maximal_Independent_Set(d_offsets, d_colIndices, numNodes, d_MIS);
    // thrust::host_vector<int> h_MIS = d_MIS; // copy the vector to host

    // // Print result
    // std::cout << "Maximal Independent Set: ";
    // for (int i = 0; i < numNodes; ++i) {
    //     if (h_MIS[i] == 1) {
    //         std::cout << i << " ";
    //     }
    // }
    // std::cout << std::endl;

    thrust::device_vector<int> d_P(numNodes, 1); // Initial vertex set P is the entire graph
    thrust::device_vector<int> StrongCompSet(numNodes, 0); // Store the strongly connected components

    Parallel_SCC_CSR(d_offsets, d_colIndices, d_P, StrongCompSet, numNodes);

    thrust::host_vector<int> h_StrongCompSet = StrongCompSet;

    std::cout << "Strongly Connected Components: ";
    for (int i = 0; i < numNodes; ++i) {
        if (h_StrongCompSet[i] == 1) {
            std::cout << i << " ";
        }
    }
    std::cout << std::endl;

    hipDeviceSynchronize();
    return 0;
}

